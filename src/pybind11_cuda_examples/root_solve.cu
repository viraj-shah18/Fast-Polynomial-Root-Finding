#include "hip/hip_runtime.h"
/*************************************************************************
/* Author: Viraj Shah
/* Email : vishah@ucsd.edu
/* University of California, San Diego
/*************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_STREAMS 4
#define THREADS_PER_BLOCK 16
#define DEFAULT_NO_ROOT 10000
#define maxIterations 100

__global__ void getRootBisection(double* low, double* high, double* out, double* poly, int degree);
__device__ double getFunctionValue(double* poly, int degree, double x);


__device__ double getFunctionValue(double* poly, int degree, double x) {
	/**
	 * @brief Finds the value of a polynomial at a given point.
	 *
	 * This function calculates the value of a polynomial at a given point x.
	 * 
	 * @param poly The polynomial to be evaluated. The polynomial is stored as an array of doubles, where the index of the array is the power of the variable.
	 * @param degree The degree of the polynomial.
	 * @param x The point at which the polynomial is evaluated.
	 * @return The value of the polynomial at the given point.
	*/
	

	// calculate value of polynomial at x
	double value = 0;
	for (int i = degree; i >= 0; i--) {
		value = value * x + poly[i];
	}

	return value;
}


__global__ void getRootBisection(double* low, double* high, double* out, double* poly, int degree) {
	// the function finds the root of a polynomial using the bisection method
	/**
	* @brief Calculates the root of a function using the bisection method.
	*
	* @param low The lower bound of the interval in which the root is to be found.
	* @param high The upper bound of the interval in which the root is to be found.
	* @param out The output array in which the root is to be stored.
	* @param poly The polynomial whose root is to be found. The polynomial is stored as an array of doubles, where the index of the array is the power of the variable.
	* @param degree The degree of the polynomial.
	* @return None
	*/
	
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

	double thread_low = low[idx];
	double thread_high = high[idx];

	double fn_low = getFunctionValue(poly, degree, thread_low);
	double fn_high = getFunctionValue(poly, degree, thread_high);

	// check for change of signs 
	//	- if no change -> add -1 and return
	if (fn_low * fn_high > 0) {
		out[idx] = DEFAULT_NO_ROOT;
		return;
	}

	if (fn_low == 0) {
		out[idx] = thread_low;
		return;
	}
	if (fn_high == 0) {
		out[idx] = thread_high;
		return;
	}

	// run while loop till convergence
	double mid = 0;
	double tol = 1e-5;
	while (abs(thread_high-thread_low) > tol) {
		mid = (thread_low + thread_high) / 2;

		if (getFunctionValue(poly, degree, thread_low) * getFunctionValue(poly, degree, mid) < 0) {
			thread_high = mid;
		}
		else {
			thread_low = mid;
		}
	}

	// add value to output
	out[idx] = mid;

	return;
}


__global__ void getRootSecant(double* low, double* high, double* h_out, double* poly, int degree) {
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	double o_low = low[idx];
	double o_high = high[idx];

	
	double curr = low[idx];
	double prev = high[idx];
	double fn_curr, fn_prev;

	double tol = 1e-5;
	int iteration = 0;
	while (iteration < maxIterations) {
		fn_curr = getFunctionValue(poly, degree, curr); 
		fn_prev = getFunctionValue(poly, degree, prev);

		if (curr < o_low || curr > o_high) {
			h_out[idx] = DEFAULT_NO_ROOT;
			return;
		}
		
		if (abs(curr - prev) < tol || abs(fn_curr) < tol || iteration >= maxIterations) {
			break;
		}
		
		double next_val = curr - fn_curr * (curr - prev) / (fn_curr - fn_prev);

		prev = curr;
		curr = next_val;

		iteration++;
	}

	h_out[idx] = curr;

}



void cu_root_solve(double low, double high, double* poly, double* h_out, int method, int degree, int num_intervals) {
	/**
	* @brief Finds the roots of a polynomial.
	*
	* This function finds the roots of a polynomial using the bisection or secant method.
	*
	* @param low The lower bound of the interval in which the root is to be found.
	* @param high The upper bound of the interval in which the root is to be found.
	* @param poly The polynomial whose root is to be found. The polynomial is stored as an array of doubles, where the index of the array is the power of the variable.
	* @param h_out array to store the output roots
	* @param degree The degree of the polynomial.
	* @param method The method to be used to find the root. The method can be either BISECTION or SECANT.
	* @return None
	**/

	double* d_low;
	double* d_high;
	double* d_out;
	double* d_poly;

	double* h_low;
	double* h_high;

	int size = sizeof(double) * num_intervals;
	int poly_size = sizeof(double) * (degree + 1);


	hipHostAlloc((void**)&h_low, size, hipHostMallocDefault);
	hipHostAlloc((void**)&h_high, size, hipHostMallocDefault);

	hipMalloc((void**)&d_low, size);
	hipMalloc((void**)&d_high, size);
	hipMalloc((void**)&d_out, size);
	hipMalloc((void**)&d_poly, poly_size);

	// creating intervals in host
	double interval_length = (high - low) / num_intervals;
	for (int i = 0; i < num_intervals; i++) {
		h_low[i] = low + i * interval_length;
		h_high[i] = low + (i + 1) * interval_length;
	}

	// create 4 cuda streams
	hipStream_t streams[NUM_STREAMS];
	int nsdata = num_intervals / NUM_STREAMS;
	size_t iBytes = nsdata * sizeof(double);
	size_t polyBytes = (degree + 1) * sizeof(double);

	dim3 block(THREADS_PER_BLOCK);
	dim3 grid((nsdata + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&streams[i]);
	}

	for (int i = 0; i < NUM_STREAMS; ++i) {
		int offset = i * nsdata;
		hipMemcpyAsync(&d_low[offset], &h_low[offset], iBytes, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&d_high[offset], &h_high[offset], iBytes, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(d_poly, poly, polyBytes, hipMemcpyHostToDevice, streams[i]);

		// run kernel code -> depending on method specified
		if (method == 0) {
			getRootBisection << <grid, block, 0, streams[i] >> > (&d_low[offset], &d_high[offset], &d_out[offset], d_poly, degree);
		}
		else if (method == 1) {
			getRootSecant <<<grid, block, 0, streams[i] >>> (&d_low[offset], &d_high[offset], &d_out[offset], d_poly, degree);
		}
		else {
			printf("Invalid method specified. Exiting.\n");
			exit(1);
		}

		hipMemcpyAsync(&h_out[offset], &d_out[offset], iBytes, hipMemcpyDeviceToHost, streams[i]);

	}

	for (int i = 0; i < NUM_STREAMS; ++i) {
		hipStreamSynchronize(streams[i]);
	}

	for (int i = 0; i < NUM_STREAMS; ++i) {
		hipStreamDestroy(streams[i]);
	}

	hipHostFree(h_low);
	hipHostFree(h_high);

	hipFree(d_low);
	hipFree(d_high);
	hipFree(d_out);
	hipFree(d_poly);
}
